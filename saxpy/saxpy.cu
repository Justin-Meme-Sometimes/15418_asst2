#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

extern float toBW(int bytes, float sec);

__global__ void saxpy_kernel(int N, float alpha, float *x, float *y,
                             float *result) {

  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < N)
    result[index] = alpha * x[index] + y[index];
}

void saxpyCuda(int N, float alpha, float *xarray, float *yarray,
               float *resultarray) {

  int totalBytes = sizeof(float) * 3 * N;

  // compute number of blocks and threads per block
  const int threadsPerBlock = 512;
  const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

  float *device_x;
  float *device_y;
  float *device_result;
  int bytes = sizeof(float) * N;

  //
  // TODO allocate device memory buffers on the GPU using hipMalloc
  //
  cudaCheckError(hipMalloc(&device_x, bytes));
  cudaCheckError(hipMalloc(&device_y, bytes));
  cudaCheckError(hipMalloc(&device_result, bytes));

  // start timing after allocation of device memory
  double startTime = CycleTimer::currentSeconds();

  //
  // TODO copy input arrays to the GPU using hipMemcpy
  //
  cudaCheckError(hipMemcpy(device_x, xarray, bytes, hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(device_y, yarray, bytes, hipMemcpyHostToDevice));

  double startTime2 = CycleTimer::currentSeconds();

  // run kernel
  saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y,
                                            device_result);
  hipDeviceSynchronize();
  double endTime2 = CycleTimer::currentSeconds();

  //
  // TODO copy result from GPU using hipMemcpy
  //
  cudaCheckError(
      hipMemcpy(resultarray, device_result, bytes, hipMemcpyDeviceToHost));
  // end timing after result has been copied back into host memory
  double endTime = CycleTimer::currentSeconds();

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode,
            hipGetErrorString(errCode));
  }

  double overallDuration = endTime - startTime;
  double overallDuration2 = endTime2 - startTime2;

  printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration,
         toBW(totalBytes, overallDuration));

  printf("Our Overall2: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration2,
         toBW(totalBytes, overallDuration2));

  // TODO free memory buffers on the GPU

  cudaCheckError(hipFree(device_x));
  cudaCheckError(hipFree(device_y));
  cudaCheckError(hipFree(device_result));
}

void printCudaInfo() {

  // for fun, just print out some stats on the machine

  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}
